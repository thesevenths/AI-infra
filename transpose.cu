#include <hip/hip_runtime.h>
#include <iostream>

__global__ void naiveGmem(float *out, float *in, int nx, int ny) {
  unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
  if (ix < nx && iy < ny) {
    out[ix * ny + iy] = in[iy * nx + ix];
  }
}

void call_naiveGmem(float *d_out, float *d_in, int nx, int ny) {
  dim3 blockSize(2, 2); 
  dim3 gridSize((nx + blockSize.x - 1) / blockSize.x,
                (ny + blockSize.y - 1) / blockSize.y);
  naiveGmem<<<gridSize, blockSize>>>(d_out, d_in, nx, ny);
}

int main() {
  int nx = 4;
  int ny = 4;
  size_t size = nx * ny * sizeof(float);


  float *h_in = (float *)malloc(size);
  float *h_out = (float *)malloc(size);

  for (int i = 0; i < nx * ny; i++) {
    h_in[i] = float(int(i) % 11);
  }

  float *d_in, *d_out;
  hipMalloc(&d_in, size);
  hipMalloc(&d_out, size);


  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

  call_naiveGmem(d_out, d_in, nx, ny);

  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {
      std::cout << h_in[j * nx + i] << " ";
    }
    std::cout << "\n";
  }

  printf("---------------\n");

  for (int j = 0; j < ny; ++j) {
    for (int i = 0; i < nx; ++i) {
      std::cout << h_out[j * nx + i] << " ";
    }
    std::cout << "\n";
  }

  free(h_in);
  free(h_out);
  hipFree(d_in);
  hipFree(d_out);

  std::cout << "Matrix transposition completed." << std::endl;

  return 0;
}